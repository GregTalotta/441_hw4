#include "hip/hip_runtime.h"
/***********************************************************************
 * sobel-cpu.cu
 *
 * Implements a Sobel filter on the image that is hard-coded in main.
 * You might add the image name as a command line option if you were
 * to use this more than as a one-off assignment.
 *
 * See https://stackoverflow.com/questions/17815687/image-processing-implementing-sobel-filter
 * or https://blog.saush.com/2011/04/20/edge-detection-with-the-sobel-operator-in-ruby/
 * for info on how the filter is implemented.
 *
 * Compile/run with:  nvcc sobel-cpu.cu -lfreeimage
 *
 ***********************************************************************/
#include "FreeImage.h"
#include "stdio.h"
#include "math.h"

// Returns the index into the 1d pixel array
// Given te desired x,y, and image width
__device__ int pixelIndex(int x, int y, int width)
{
  return (y * width + x);
}

// Returns the sobel value for pixel x,y
__global__ void sobel(int width, char *pixels, int *c)
{
  int x = blockIdx.y;
  int y = blockIdx.x;
  // ignore edges
  if(x > 0 && y > 0 && x < blockDim.x-1 && y < blockDim.y-1)
  {
    int x00 = -1;
    int x20 = 1;
    int x01 = -2;
    int x21 = 2;
    int x02 = -1;
    int x22 = 1;
    x00 *= pixels[pixelIndex(x - 1, y - 1, width)];
    x01 *= pixels[pixelIndex(x - 1, y, width)];
    x02 *= pixels[pixelIndex(x - 1, y + 1, width)];
    x20 *= pixels[pixelIndex(x + 1, y - 1, width)];
    x21 *= pixels[pixelIndex(x + 1, y, width)];
    x22 *= pixels[pixelIndex(x + 1, y + 1, width)];

    int y00 = -1;
    int y10 = -2;
    int y20 = -1;
    int y02 = 1;
    int y12 = 2;
    int y22 = 1;
    y00 *= pixels[pixelIndex(x - 1, y - 1, width)];
    y10 *= pixels[pixelIndex(x, y - 1, width)];
    y20 *= pixels[pixelIndex(x + 1, y - 1, width)];
    y02 *= pixels[pixelIndex(x - 1, y + 1, width)];
    y12 *= pixels[pixelIndex(x, y + 1, width)];
    y22 *= pixels[pixelIndex(x + 1, y + 1, width)];

    int px = x00 + x01 + x02 + x20 + x21 + x22;
    int py = y00 + y10 + y20 + y02 + y12 + y22;
    c[pixelIndex(x, y, width)] = (int)sqrtf(px * px + py * py);
  }
  else
  {
    c[pixelIndex(x, y, width)] = pixels[pixelIndex(x, y, width)];
  }
}

int main()
{
  FreeImage_Initialise();
  atexit(FreeImage_DeInitialise);

  // Load image and get the width and height
  FIBITMAP *image;
  image = FreeImage_Load(FIF_PNG, "coins.png", 0);
  if (image == NULL)
  {
    printf("Image Load Problem\n");
    exit(0);
  }
  int imgWidth;
  int imgHeight;
  imgWidth = FreeImage_GetWidth(image);
  imgHeight = FreeImage_GetHeight(image);

  int *c;
  int *dev_c;
  c = (int *)malloc(sizeof(int) * imgWidth * imgHeight);
  hipMalloc((void **)&dev_c, sizeof(int) * imgWidth * imgHeight);

  // Convert image into a flat array of chars with the value 0-255 of the
  // greyscale intensity
  RGBQUAD aPixel;
  char *pixels;
  int pixIndex = 0;
  pixels = (char *)malloc(sizeof(char) * imgWidth * imgHeight);
  for (int i = 0; i < imgHeight; i++)
    for (int j = 0; j < imgWidth; j++)
    {
      FreeImage_GetPixelColor(image, j, i, &aPixel);
      char grey = ((aPixel.rgbRed + aPixel.rgbGreen + aPixel.rgbBlue) / 3);
      pixels[pixIndex++] = grey;
    }

  char *dev_pixels;
  hipMalloc((void **)&dev_pixels, sizeof(char) * imgWidth * imgHeight);
  hipMemcpy(dev_pixels, pixels, sizeof(char) * imgWidth * imgHeight, hipMemcpyHostToDevice);


  // ** change
  dim3 threadsPerBlock(1);             //one thred per block
  dim3 numBlocks(imgHeight, imgWidth); // one block per pixel
  printf("height is: %d\n", imgHeight);
  printf("width is: %d\n", imgWidth);
  sobel<<<numBlocks, threadsPerBlock>>>(imgWidth, dev_pixels, dev_c);
  hipMemcpy(c, dev_c, sizeof(int) * imgWidth * imgHeight, hipMemcpyDeviceToHost);


  // Apply sobel operator to pixels, ignoring the borders ** change to use c arr
  FIBITMAP *bitmap = FreeImage_Allocate(imgWidth, imgHeight, 24);
  for (int i = 1; i < imgWidth - 1; i++)
  {
    for (int j = 1; j < imgHeight - 1; j++)
    {
      int sVal = c[j*imgWidth + i]; //change to use arr answers
      aPixel.rgbRed = sVal;
      aPixel.rgbGreen = sVal;
      aPixel.rgbBlue = sVal;
      FreeImage_SetPixelColor(bitmap, i, j, &aPixel);
    }
  }
  FreeImage_Save(FIF_PNG, bitmap, "coins-edge.png", 0);

  free(pixels);
  FreeImage_Unload(bitmap);
  FreeImage_Unload(image);
  return 0;
}
